#include "hip/hip_runtime.h"
#include <ATen/Dispatch.h>
#include <ATen/ExpandUtils.h>
#include <ATen/NativeFunctions.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/AccumulateType.h>
#include <ATen/CUDAGenerator.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_kernel.h>
#include <utility>
#include <functional>

#include <ATen/native/Distributions.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/TensorIterator.h>
#include <ATen/LegacyTHFunctionsCUDA.h>

#include <THC/THCGeneral.h>
#include <THC/THCTensorRandom.h>
#include <THC/THCGenerator.hpp>
#include <THC/THCApply.cuh>
#include <THC/THCDeviceUtils.cuh>

#include <cstdint>
#include <limits>
#include <utility>
#include <type_traits>

/**
 * Note [Register spilling in hiprand call for CUDA < 10]
 * ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
 * For CUDA < 10, hiprandStatePhilox4_32_10_t engine achieves poor performance (60% SOL bandwidth)
 * when called to generate one random number at a time. This is because the line
 *            unsigned ret = (&state->output.x)[state->STATE++];
 * in
 *            QUALIFIERS unsigned int hiprand(hiprandStatePhilox4_32_10_t *state)
 * in hiprand/hiprand_kernel.h dynamically indexes into state.output, preventing the compiler from ever
 * storing state.output in registers.
 *
 * CUDA 10 fixed this problem. However, for backwards compatibility, in the following kernels
 * we are using hiprand distributions that utilize curand4 call. curand4 call doesn't have the
 * register spilling problem.
 */
 
THCGenerator* THCRandom_getGenerator(THCState* state);

namespace {
// Increment should be at least the number of hiprand() random numbers used in
// each thread. It is the user's responsibility to make sure that the increment for philox is never
// smaller than the number of hiprand() calls. Increment value > the number of hiprand() calls
// won't harm but anything less would mean that you would be reusing random values from
// previous calls. 
// e.g. In many kernels below, we use distributions that utilize curand4 call in the kernel.
//      Hence, increment value should be at least 4 for those kernels.
std::pair<uint64_t, uint64_t> next_philox_seed(at::Generator* gen, uint64_t increment) {
  auto gen_ = THCRandom_getGenerator(at::globalContext().getTHCState());
  uint64_t offset = gen_->state.philox_seed_offset.fetch_add(increment);
  return std::make_pair(gen_->state.initial_seed, offset);
}

// launch bounds used for kernels utilizing TensorIterator
const uint32_t block_size_bound = 256;
const uint32_t grid_size_bound = 4;
// number of randoms given by distributions like hiprand_uniform4, hiprand_uniform2_double
// used in calculating philox offset.
const uint32_t curand4_engine_calls = 4;

// utility function that calculates proper philox_offset
// for distributions utilizing TensorIterator. For distributions using
// TensorIterator, we are using a grid-stride loop with each
// thread yielding one element per thread. For the edge of the grid-stride
// loop, if the tensor size is large, the unroll loop will kick in and the float4
// from curand4 will start getting utilized (for common tensor sizes, we end up
// using rand.x from each thread). Hence, the philox_offset is 
// (number of elements per thread * number of engine calls), which makes
// sure that philox offset increment is not less than the number of randoms used
// in each thread.
std::tuple<uint64_t, dim3, dim3> calc_execution_policy(int64_t total_elements) {
  const uint64_t numel = static_cast<uint64_t>(total_elements);
  const uint32_t block_size = block_size_bound;
  const uint32_t unroll = curand4_engine_calls;
  dim3 dim_block(block_size);
  dim3 grid((numel + block_size - 1) / block_size);
  uint32_t blocks_per_sm = at::cuda::getCurrentDeviceProperties()->maxThreadsPerMultiProcessor / block_size;
  grid.x = std::min(
      static_cast<uint32_t>(at::cuda::getCurrentDeviceProperties()->multiProcessorCount) * blocks_per_sm,
      grid.x);
  //number of times random will be generated per thread, to offset philox counter in thc random state
  uint64_t counter_offset = ((numel - 1) / (block_size * grid.x * unroll) + 1)
                                * curand4_engine_calls;
  return std::make_tuple(counter_offset, grid, dim_block);
}

// grid stride loop kernel for distributions
template<typename accscalar_t, int unroll_factor, typename dist_t, typename transform_t>
C10_LAUNCH_BOUNDS_2(block_size_bound, grid_size_bound)
__global__ void distribution_elementwise_grid_stride_kernel(int numel,
                                                            std::pair<uint64_t, uint64_t> seeds,
                                                            const dist_t dist_func,
                                                            const transform_t transform_func) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandStatePhilox4_32_10_t state;
  hiprand_init(
      seeds.first,
      idx,
      seeds.second,
      &state);
  int rounded_size = ((numel - 1)/(blockDim.x * gridDim.x * unroll_factor)+1) *
      blockDim.x * gridDim.x * unroll_factor;
  for(int linear_index = idx; linear_index < rounded_size; linear_index += blockDim.x * gridDim.x * unroll_factor) {
    auto rand = dist_func(&state);
    #pragma unroll
    for (int ii = 0; ii < unroll_factor; ii++) {
      int li = linear_index + blockDim.x * gridDim.x * ii;
      if (li < numel) {
        transform_func(li, static_cast<accscalar_t>((&rand.x)[ii]));
      }
    }
    __syncthreads(); 
  }
}

/**
 * distribution_nullary_kernel is analogous to gpu_nullary_kernel in
 * ATen/native/cuda/Loops.cuh. Like gpu_nullary_kernel, it uses
 * TensorIterator to launch a kernel. However, the differences are
 *   - it launches a grid-stride loop based kernel. The kernel is not
 *     generic like elementwise_kernel in Loops.cuh and is specialized
 *     for the distribution kernels here.
 *   - For big size tensors, we can launch multiple kernels recursively
 *     (i.e. if (!iter.can_use_32bit_indexing())) and hence, the philox
 *     offset calculation is done in this function.
 *
 * FIXME: Can we specialize elementwise_kernel and launch_kernel in Loops.cuh
 * to have grid-stride loop kernel and then use that to launch our distribution
 * kernels? Note that we need a grid-stride loop kernel because, we found by testing
 * that it achieves peak effective bandwidth.
 */
template<typename scalar_t, 
         typename accscalar_t,
         int unroll_factor,
         typename dist_t,
         typename transform_t>
void distribution_nullary_kernel(at::TensorIterator& iter,
                                 at::Generator* gen,
                                 const dist_t& dist_func,
                                 const transform_t transform_func) {
  static_assert(unroll_factor >= 1, "unroll_factor must be >= 1.");
  int64_t numel = iter.numel();
  if (numel == 0) {
    return;
  }
  
  auto execution_policy = calc_execution_policy(numel);
  auto counter_offset = std::get<0>(execution_policy);
  auto grid = std::get<1>(execution_policy);
  auto block = std::get<2>(execution_policy);
  auto seeds = next_philox_seed(gen, counter_offset);

  if (!iter.can_use_32bit_indexing()) {
    for (auto& sub_iter : iter.with_32bit_indexing()) {
      distribution_nullary_kernel<scalar_t, accscalar_t, unroll_factor>(sub_iter,
        gen, dist_func, transform_func);
    }
    return;
  }

  char* out_data = (char*)iter.data_ptr(0);

  auto stream = at::cuda::getCurrentCUDAStream();
  if (iter.is_trivial_1d()) {
    auto strides = iter.get_inner_strides();
    int stride0 = strides[0];
    distribution_elementwise_grid_stride_kernel<accscalar_t, unroll_factor><<<grid, block, 0, stream>>>(
      numel,
      seeds,
      dist_func,
      [=]__device__(int idx, accscalar_t rand) {
        scalar_t* out = (scalar_t*)&out_data[stride0 * idx];
        *out = transform_func(rand);
      }
    );
  } else {
    auto offset_calc = at::native::make_offset_calculator<1>(iter);
    distribution_elementwise_grid_stride_kernel<accscalar_t, unroll_factor><<<grid, block, 0, stream>>>(
      numel,
      seeds,
      dist_func,
      [=]__device__(int idx, accscalar_t rand) {
        auto offsets = offset_calc.get(idx);
        scalar_t* out = (scalar_t*)&out_data[offsets[0]];
        *out = transform_func(rand);
      }
    );
  }
  AT_CUDA_CHECK(hipGetLastError());
}

template <typename scalar_t>
void poisson_cuda_kernel(
    at::Tensor& ret,
    const at::Tensor& lambda,
    std::pair<uint64_t, uint64_t> seeds) {
  at::cuda::CUDA_tensor_apply2<scalar_t, scalar_t>(
      ret,
      lambda,
      [seeds] __device__(
          scalar_t & ret_val, const scalar_t& lambda) {
        hiprandStatePhilox4_32_10_t state;
        hiprand_init(
            seeds.first,
            blockIdx.x * blockDim.x + threadIdx.x,
            seeds.second,
            &state);
        ret_val = static_cast<scalar_t>(hiprand_poisson(&state, lambda));
      });
}

template <typename scalar_t>
void gamma_cuda_kernel(
    at::Tensor& ret,
    const at::Tensor& alpha,
    std::pair<uint64_t, uint64_t> seeds) {
  using accscalar_t = at::acc_type<scalar_t, true>;
  at::cuda::CUDA_tensor_apply2<scalar_t, scalar_t>(
      ret,
      alpha,
      [seeds] __device__(
          scalar_t & ret_val, const scalar_t& alpha) {
        hiprandStatePhilox4_32_10_t state;
        hiprand_init(
            seeds.first,
            blockIdx.x * blockDim.x + threadIdx.x,
            seeds.second,
            &state);

        auto uniform_lambda = [&state] __device__ () {
          return hiprand_uniform(&state);
        };
        BaseSampler<accscalar_t, decltype(uniform_lambda)> standard_uniform(uniform_lambda);

        auto normal_lambda = [&state] __device__ () {
          return hiprand_normal(&state);
        };
        BaseSampler<accscalar_t, decltype(normal_lambda)> standard_normal(normal_lambda);
        auto sample = sample_gamma<scalar_t, accscalar_t, decltype(uniform_lambda), decltype(normal_lambda)>(alpha, standard_uniform, standard_normal);
        auto min_value = std::numeric_limits<scalar_t>::min();
        ret_val = (min_value > sample) ? min_value : sample;
      });
}

template <typename scalar_t>
void gamma_grad_cuda_kernel(
    at::Tensor& ret,
    const at::Tensor& self,
    const at::Tensor& output) {
  using accscalar_t = at::acc_type<scalar_t, true>;
  at::cuda::CUDA_tensor_apply3<scalar_t, scalar_t, scalar_t>(
      ret, self, output,
      [] __device__ (scalar_t& ret_val, const scalar_t& self_val, const scalar_t &output_val) {
        ret_val = standard_gamma_grad_one<scalar_t, accscalar_t>(self_val, output_val);
      });
}

template<typename scalar_t, typename prob_t>
void bernoulli_tensor_cuda_kernel(
    at::Tensor& ret, const at::Tensor& p,
    std::pair<uint64_t, uint64_t> seeds) {
  // The template argument `4` below indicates that we want to operate on four
  // element at each time. See NOTE [ CUDA_tensor_applyN helpers ] for details.
  at::cuda::CUDA_tensor_apply2<scalar_t, prob_t, 4>(
      ret, p,
      [seeds] __device__(
          int n, scalar_t& v1, scalar_t& v2, scalar_t& v3, scalar_t& v4,
          const prob_t& p1, const prob_t& p2, const prob_t& p3, const prob_t& p4) {
        hiprandStatePhilox4_32_10_t state;
        hiprand_init(
            seeds.first,
            blockIdx.x * blockDim.x + threadIdx.x,
            seeds.second,
            &state);
        // See Note [Register spilling in hiprand call for CUDA < 10]
        float4 rand = hiprand_uniform4(&state);
        switch (n) {
          case 4: {
            assert(0 <= p4 && p4 <= 1);
            v4 = static_cast<scalar_t>(rand.w <= p4);
            // fallthrough
          }
          case 3: {
            assert(0 <= p3 && p3 <= 1);
            v3 = static_cast<scalar_t>(rand.z <= p3);
            // fallthrough
          }
          case 2: {
            assert(0 <= p2 && p2 <= 1);
            v2 = static_cast<scalar_t>(rand.y <= p2);
            // fallthrough
          }
          case 1: {
            assert(0 <= p1 && p1 <= 1);
            v1 = static_cast<scalar_t>(rand.x <= p1);
          }
        }
      }
    );
}

template<typename scalar_t>
void bernoulli_scalar_cuda_kernel(
    at::Tensor& ret, double p_,
    std::pair<uint64_t, uint64_t> seeds) {
  float p = static_cast<float>(p_);
  // The template argument `4` below indicates that we want to operate on four
  // element at each time. See NOTE [ CUDA_tensor_applyN helpers ] for details.
  at::cuda::CUDA_tensor_apply1<scalar_t, 4>(
      ret, [seeds, p] __device__(
        int n, scalar_t& v1, scalar_t& v2, scalar_t& v3, scalar_t& v4) {
        hiprandStatePhilox4_32_10_t state;
        hiprand_init(
            seeds.first,
            blockIdx.x * blockDim.x + threadIdx.x,
            seeds.second,
            &state);
        // See Note [Register spilling in hiprand call for CUDA < 10]
        float4 rand = hiprand_uniform4(&state);
        switch (n) {
          case 4: {
            v4 = static_cast<scalar_t>(rand.w <= p);
            // fallthrough
          }
          case 3: {
            v3 = static_cast<scalar_t>(rand.z <= p);
            // fallthrough
          }
          case 2: {
            v2 = static_cast<scalar_t>(rand.y <= p);
            // fallthrough
          }
          case 1: {
            v1 = static_cast<scalar_t>(rand.x <= p);
          }
        }
      }
    );
}

template<typename scalar_t>
void dirichlet_scalar_cuda_kernel(
    at::Tensor& ret,
    const at::Tensor& gamma) {
  auto gamma_sum = gamma.sum(-1, true).expand(ret.sizes());
  at::cuda::CUDA_tensor_apply3<scalar_t, scalar_t, scalar_t>(ret, gamma, gamma_sum,
  [] __device__(scalar_t &ret_val, const scalar_t &gamma, const scalar_t &gamma_sum) {
    ret_val = gamma / gamma_sum;
    auto min_value = std::numeric_limits<scalar_t>::min();
    auto max_value = 1 - std::numeric_limits<scalar_t>::epsilon();
    ret_val = (min_value > ret_val) ? min_value : ret_val;
    ret_val = (max_value < ret_val) ? max_value : ret_val;
  });
}

} // namespace

namespace at { namespace native {
Tensor _s_poisson_cuda(const Tensor& lambda, Generator* gen) {
  Tensor ret = at::empty(lambda.sizes(), lambda.options());
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(ret.scalar_type(), "poisson_cuda", [&] {
    poisson_cuda_kernel<scalar_t>(ret, lambda, next_philox_seed(gen, 20));
  });
  return ret;
}

Tensor _s_gamma_cuda(const Tensor& alpha, Generator* gen) {
  Tensor ret = at::empty(alpha.sizes(), alpha.options());
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(ret.scalar_type(), "gamma_cuda", [&] {
     gamma_cuda_kernel<scalar_t>(ret, alpha, next_philox_seed(gen, 10));
   });
  return ret;
}

Tensor _s_dirichlet_cuda(const Tensor& alpha, Generator* gen) {
  Tensor ret = at::empty(alpha.sizes(), alpha.options());
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(ret.scalar_type(), "dirichlet", [&] {
    Tensor gamma = at::empty(alpha.sizes(), alpha.options());
    gamma_cuda_kernel<scalar_t>(gamma, alpha, next_philox_seed(gen, 10));
    dirichlet_scalar_cuda_kernel<scalar_t>(ret, gamma);
  });
  return ret;
}

Tensor _standard_gamma_grad_cuda(const Tensor& self, const Tensor& output) {
  Tensor ret = at::empty(self.sizes(), self.options());
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(self.scalar_type(), "_standard_gamma_grad_cuda", [&] {
     gamma_grad_cuda_kernel<scalar_t>(ret, self, output);
   });
  return ret;
}

Tensor& bernoulli_tensor_cuda_(Tensor &self, const Tensor& p_, Generator* gen) {
  auto p = std::get<0>(expand_inplace(self, p_.to(kCUDA)));
  AT_DISPATCH_ALL_TYPES_AND(
    at::ScalarType::Half, self.scalar_type(), "bernoulli_tensor_cuda_self_", [&] {
      using self_t = scalar_t;
      auto seeds = next_philox_seed(gen, 10);
      AT_DISPATCH_FLOATING_TYPES_AND_HALF(p.scalar_type(), "bernoulli_tensor_cuda_p_", [&] {
        using p_t = scalar_t;
        return bernoulli_tensor_cuda_kernel<self_t, p_t>(self, p, seeds);
      });
   });
  return self;
}

Tensor& bernoulli_scalar_cuda_(Tensor &self, double p, Generator* gen) {
  TORCH_CHECK(0 <= p && p <= 1, "bernoulli_ expects p to be in [0, 1], but got p=", p);
  AT_DISPATCH_ALL_TYPES_AND(at::ScalarType::Half, self.scalar_type(), "bernoulli_scalar_cuda_", [&] {
    auto seeds = next_philox_seed(gen, 10);
    bernoulli_scalar_cuda_kernel<scalar_t>(self, p, seeds);
   });
  return self;
}

void uniform_kernel_cuda(TensorIterator& iter, double from_, double to_, Generator* gen_) {
  auto gen = check_generator<CUDAGenerator>(gen_, &globalContext().defaultGenerator(kCUDA));
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "uniform_cuda", [&] {
    auto from = static_cast<scalar_t>(from_);
    auto to = static_cast<scalar_t>(to_);
    TORCH_CHECK(from <= to,
      "uniform_ expects to return a [from, to) range, but found from=", from,
      " > to=", to);
    TORCH_CHECK((to - from) <= std::numeric_limits<scalar_t>::max(),
          "uniform_ expects to-from <= std::numeric_limits<", toString(iter.dtype()),
          ">::max(), but found to=", to, " and from=", from,
          " which result in to-from to exceed the limit");

    using accscalar_t = at::acc_type<scalar_t, true>;
    auto range = static_cast<accscalar_t>(to-from);
    from = static_cast<accscalar_t>(from);
    // define lambda to reverse bounds, multiply 'range' and add 'from_'
    auto uniform_func = [range, from] __device__ (accscalar_t rand) {
      // reverse the bounds of curand4 from (0, 1] to [0, 1)
      // Note that this method is from legacy THCTensorRandom and is likely to give
      // you more 0-s, since, the probability of gettings 1-s is higher than 0-s and
      // by reversing the bounds, we are flipping the probabilities of 1-s and 0-s.
      auto reverse_bound_rand = rand == static_cast<accscalar_t>(1.0) ? static_cast<accscalar_t>(0.0) : rand;
      return static_cast<scalar_t>(reverse_bound_rand * range + from);
    };
    if (std::is_same<scalar_t, double>::value) {
      distribution_nullary_kernel<scalar_t, accscalar_t, curand4_engine_calls/2>(iter,
        gen,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) { return hiprand_uniform2_double(state); },
        uniform_func);
    } else {
      distribution_nullary_kernel<scalar_t, accscalar_t, curand4_engine_calls>(iter,
        gen,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) { return hiprand_uniform4(state); },
        uniform_func);
    }
   });
}

void random_kernel_cuda(TensorIterator& iter, uint64_t range, int64_t base, Generator* gen_) {
  auto gen = check_generator<CUDAGenerator>(gen_, &globalContext().defaultGenerator(kCUDA));
  AT_DISPATCH_ALL_TYPES_AND2(at::ScalarType::Bool, at::ScalarType::Half, iter.dtype(), "random_cuda", [&] {
    if (std::is_same<scalar_t, double>::value || std::is_same<scalar_t, int64_t>::value) {
      // define lambda to mod with range and add base
      auto random_func = [range, base] __device__ (uint64_t rand) {
        return static_cast<int64_t>(rand % range + base);
      };
      distribution_nullary_kernel<scalar_t, uint64_t, curand4_engine_calls/2>(iter,
        gen,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) -> ulonglong2 {
          ulonglong2 ret;
          uint4 rand_val = curand4(state);
          ret.x = (static_cast<uint64_t>(rand_val.x) << 32) | rand_val.y;
          ret.y = (static_cast<uint64_t>(rand_val.z) << 32) | rand_val.w;
          return ret;
        },
        random_func);
    } else {
      auto random_func = [range, base] __device__ (uint32_t rand) {
        return static_cast<int32_t>(rand % static_cast<uint32_t>(range) + static_cast<int32_t>(base));
      };
      distribution_nullary_kernel<scalar_t, uint32_t, curand4_engine_calls>(iter,
        gen,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) {
          return curand4(state);
        },
        random_func);
    }
   });
}

void normal_kernel_cuda(TensorIterator& iter, double mean_, double std_, Generator* gen_) {
  auto gen = check_generator<CUDAGenerator>(gen_, &globalContext().defaultGenerator(kCUDA));
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "normal_cuda", [&] {
    using accscalar_t = at::acc_type<scalar_t, true>;
    auto mean = static_cast<accscalar_t>(mean_);
    auto std = static_cast<accscalar_t>(std_);
    // define lambda to multiply std and add mean
    auto normal_func = [mean, std] __device__ (accscalar_t rand) {
      return static_cast<scalar_t>(rand * std + mean);
    };
    if (std::is_same<scalar_t, double>::value) {
      distribution_nullary_kernel<scalar_t, accscalar_t, curand4_engine_calls/2>(iter,
        gen,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) { return hiprand_normal2_double(state); },
        normal_func);
    } else {
      distribution_nullary_kernel<scalar_t, accscalar_t, curand4_engine_calls>(iter,
        gen,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) { return hiprand_normal4(state); },
        normal_func);
    }
   });
}

Tensor& uniform_cuda_(Tensor& self, double from, double to, Generator* gen) {
  auto iter = TensorIterator::nullary_op(self);
  uniform_kernel_cuda(*iter, from, to, gen);
  return self;
}

Tensor& random_cuda_(Tensor& self, Generator* gen) {
  auto iter = TensorIterator::nullary_op(self);
  uint64_t range;
  auto iter_scalar_type = iter->dtype();
  if (isFloatingType(iter_scalar_type)) {
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter_scalar_type, "random_cuda_range_calc", [&] {
      range = static_cast<uint64_t>((1ULL << std::numeric_limits<scalar_t>::digits) + 1);
    });
  } else {
    AT_DISPATCH_INTEGRAL_TYPES(iter_scalar_type, "random_cuda_range_calc", [&] {
      range = static_cast<uint64_t>(std::numeric_limits<scalar_t>::max()) + 1;
    });
  }
  random_kernel_cuda(*iter, range, 0, gen);
  return self;
}

Tensor& clamped_random_cuda_(Tensor& self, int64_t from, int64_t to, Generator* gen) {
  TORCH_CHECK(from < to, "random_ expects 'from' to be less than 'to', but got from=", from, " >= to=", to);
  auto iter = TensorIterator::nullary_op(self);
  uint64_t range = to - from;
  random_kernel_cuda(*iter, range, from, gen);
  return self;
}

Tensor& capped_random_cuda_(Tensor& self, int64_t to, Generator* gen) {
  return clamped_random_cuda_(self, 0, to, gen);
}

Tensor& normal_cuda_(Tensor& self, double mean, double std, Generator* gen) {
  TORCH_CHECK(std > 0.0, "normal_ expects std > 0.0, but found std=", std);
  auto iter = TensorIterator::nullary_op(self);
  normal_kernel_cuda(*iter, mean, std, gen);
  return self;
}

Tensor& normal_out_cuda(Tensor& output, const Tensor& mean, double std, Generator* gen) {
  normal_cuda_(output, 0, std, gen);
  output.add_(mean);
  return output;
}

Tensor& normal_out_cuda(Tensor& output, double mean, const Tensor& std, Generator* gen) {
  normal_cuda_(output, 0, 1, gen);
  auto mean_tensor = at::full({1}, mean, output.options());
  at::native::legacy::cuda::_th_addcmul_out(output, mean_tensor, output, std, 1);
  return output;
}

Tensor& normal_out_cuda(Tensor& output, const Tensor& mean, const Tensor& std, Generator* gen) {
  normal_cuda_(output, 0, 1, gen);
  at::native::legacy::cuda::_th_addcmul_out(output, mean, output, std, 1);
  return output; 
}

Tensor normal_cuda(const Tensor& mean, double std, Generator* gen) {
  Tensor ret = at::empty(mean.sizes(), mean.options());
  normal_out_cuda(ret, mean, std, gen);
  return ret;
}

Tensor normal_cuda(double mean, const Tensor& std, Generator* gen) {
  Tensor ret = at::empty(std.sizes(), std.options());
  normal_out_cuda(ret, mean, std, gen);
  return ret;
}

Tensor normal_cuda(const Tensor& mean, const Tensor& std, Generator* gen) {
  Tensor ret = at::empty(mean.sizes(), mean.options());
  normal_out_cuda(ret, mean, std, gen);
  return ret;
}

}} // namespace at::native
