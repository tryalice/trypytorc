#include "hip/hip_runtime.h"
#include <THC/THCTensorRandom.h>
#include <THC/THCDeviceUtils.cuh>
#include <THC/THCGeneral.h>
#include <THC/THCTensorCopy.h>
#include <THC/THCTensorMath.h>
#include <THC/THCReduceApplyUtils.cuh>
#include <THC/THCTensorRandom.cuh>
#include <THC/THCGenerator.hpp>
#include <ATen/Config.h>

#include <ATen/cuda/_curand_mtgp32_host.h>

#include <thrust/functional.h>

#define MAX_NUM_BLOCKS 200
#define BLOCK_SIZE 256


THCGenerator* THCRandom_getGenerator(THCState* state);

/* Sets up generator. Allocates but does not create the generator states. Not thread-safe. */
__host__ void initializeGenerator(THCState *state, THCGenerator* gen)
{
  gen->state.gen_states = static_cast<hiprandStateMtgp32*>(THCudaMalloc(state, MAX_NUM_BLOCKS * sizeof(hiprandStateMtgp32)));
  gen->state.kernel_params = static_cast<mtgp32_kernel_params*>(THCudaMalloc(state, sizeof(mtgp32_kernel_params)));
}

/* Creates a new generator state given the seed. Not thread-safe. */
__host__ void createGeneratorState(THCGenerator* gen, uint64_t seed)
{
  if (hiprandMakeMTGP32Constants(mtgp32dc_params_fast_11213, gen->state.kernel_params) != HIPRAND_STATUS_SUCCESS)
  {
    THError("Creating MTGP constants failed.");
  }
  if (hiprandMakeMTGP32KernelState(gen->state.gen_states, mtgp32dc_params_fast_11213,
                                  gen->state.kernel_params, MAX_NUM_BLOCKS, seed) != HIPRAND_STATUS_SUCCESS)
  {
    THError("Creating MTGP kernel state failed.");
  }
  // seed and offset for philox
  gen->state.initial_seed = seed;
  gen->state.philox_seed_offset = 0;
}

THC_API __host__ void THCRandom_getRNGState(THCState* state, THByteTensor *rng_state)
{
  THCGenerator* gen = THCRandom_getGenerator(state);
  std::lock_guard<std::mutex> lock(gen->mutex);

  // The RNG state comprises the MTPG32 states, the seed, and an offset used for Philox
  static const size_t states_size = MAX_NUM_BLOCKS * sizeof(hiprandStateMtgp32);
  static const size_t seed_size = sizeof(gen->state.initial_seed);
  static const size_t offset_size = sizeof(gen->state.philox_seed_offset);
  static const size_t total_size = states_size + seed_size + offset_size;
  THByteTensor_resize1d(rng_state, total_size);
  THArgCheck(THByteTensor_nElement(rng_state) == total_size, 1, "RNG state is wrong size");
  THArgCheck(THByteTensor_isContiguous(rng_state), 1, "RNG state must be contiguous");
  THCudaCheck(hipMemcpy(THByteTensor_data(rng_state), gen->state.gen_states,
                         states_size, hipMemcpyDeviceToHost));
  memcpy(THByteTensor_data(rng_state) + states_size, &gen->state.initial_seed, seed_size);
  memcpy(THByteTensor_data(rng_state) + states_size + seed_size, &gen->state.philox_seed_offset, offset_size);
}

__global__ void set_rngstate_kernel(hiprandStateMtgp32 *state, mtgp32_kernel_params *kernel)
{
#ifndef __HIP_PLATFORM_HCC__
  state[threadIdx.x].k = kernel;
#else
  state[threadIdx.x].set_params(kernel);
#endif
}

THC_API __host__ void THCRandom_setRNGState(THCState* state, THByteTensor *rng_state)
{
  THCGenerator* gen = THCRandom_getGenerator(state);
  std::lock_guard<std::mutex> lock(gen->mutex);

  static const size_t states_size = MAX_NUM_BLOCKS * sizeof(hiprandStateMtgp32);
  static const size_t seed_size = sizeof(gen->state.initial_seed);
  static const size_t offset_size = sizeof(gen->state.philox_seed_offset);
  static const size_t total_size = states_size + seed_size + offset_size;
  bool no_philox_seed = false;
  if (THByteTensor_nElement(rng_state) == total_size - offset_size) {
    no_philox_seed = true;
  }
  else {
    THArgCheck(THByteTensor_nElement(rng_state) == total_size, 1, "RNG state is wrong size");
  }
  THArgCheck(THByteTensor_isContiguous(rng_state), 1, "RNG state must be contiguous");

  THCudaCheck(hipMemcpy(gen->state.gen_states, THByteTensor_data(rng_state),
                         states_size, hipMemcpyHostToDevice));
  set_rngstate_kernel<<<1, MAX_NUM_BLOCKS, 0, THCState_getCurrentStream(state)>>>(
      gen->state.gen_states, gen->state.kernel_params);
  memcpy(&gen->state.initial_seed, THByteTensor_data(rng_state) + states_size, seed_size);
  if (!no_philox_seed) {
    memcpy(&gen->state.philox_seed_offset, THByteTensor_data(rng_state) + states_size + seed_size, offset_size);
  }
  else {
    gen->state.philox_seed_offset = 0;
  }
}

#define GENERATE_KERNEL1(NAME, T, ARG1, CURAND_T, CURAND_FUNC, TRANSFORM)      \
__global__ void NAME(hiprandStateMtgp32 *state, int size, T *result, ARG1)    \
{                                                                              \
  int idx = blockIdx.x * BLOCK_SIZE + threadIdx.x;                             \
  int rounded_size = THCCeilDiv(size, BLOCK_SIZE) * BLOCK_SIZE;                \
  for (int i = idx; i < rounded_size; i += BLOCK_SIZE * MAX_NUM_BLOCKS) {      \
    CURAND_T x = CURAND_FUNC(&state[blockIdx.x]);                              \
    if (i < size) {                                                            \
      T y = TRANSFORM;                                                         \
      result[i] = y;                                                           \
    }                                                                          \
  }                                                                            \
}

#define GENERATE_KERNEL2(NAME, T, ARG1, ARG2, CURAND_T, CURAND_FUNC, TRANSFORM)      \
__global__ void NAME(hiprandStateMtgp32 *state, int size, T *result, ARG1, ARG2)    \
{                                                                                    \
  int idx = blockIdx.x * BLOCK_SIZE + threadIdx.x;                                   \
  int rounded_size = THCCeilDiv(size, BLOCK_SIZE) * BLOCK_SIZE;                      \
  for (int i = idx; i < rounded_size; i += BLOCK_SIZE * MAX_NUM_BLOCKS) {            \
    CURAND_T x = CURAND_FUNC(&state[blockIdx.x]);                                    \
    if (i < size) {                                                                  \
      T y = TRANSFORM;                                                               \
      result[i] = y;                                                                 \
    }                                                                                \
  }                                                                                  \
}

GENERATE_KERNEL1(generate_exponential, float, double lambda, float, hiprand_uniform, (float)(-1. / lambda * log(x)))
GENERATE_KERNEL1(generate_exponential, double, double lambda, double, hiprand_uniform_double, (double)(-1. / lambda * log(x)))

GENERATE_KERNEL2(generate_cauchy, float, double median, double sigma, float, hiprand_uniform, (float)(median + sigma * tan(M_PI*(x-0.5))))
GENERATE_KERNEL2(generate_cauchy, double, double median, double sigma, double, hiprand_uniform_double, (double)(median + sigma * tan(M_PI*(x-0.5))))

GENERATE_KERNEL1(generate_exponential, at::Half, double lambda, float, hiprand_uniform, (ScalarConvert<float, at::Half>::to((float)(-1. / lambda * log(x)))))
GENERATE_KERNEL2(generate_cauchy, at::Half, double median, double sigma, float, hiprand_uniform, (ScalarConvert<float, at::Half>::to((float)(median + sigma * tan(M_PI*(x-0.5))))))

#include <THC/generic/THCTensorRandom.cu>
#include <THC/THCGenerateAllTypes.h>

#include <THC/generic/THCTensorRandom.cu>
#include <THC/THCGenerateBoolType.h>

#undef GENERATE_KERNEL1
#undef GENERATE_KERNEL2
