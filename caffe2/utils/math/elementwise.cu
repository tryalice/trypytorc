#include "hip/hip_runtime.h"
#include "caffe2/utils/math/elementwise.h"

#include <thrust/functional.h>

#include "caffe2/core/context_gpu.h"
#include "caffe2/utils/conversions.h"
#include "caffe2/utils/math/half_utils.h"
#include "caffe2/utils/math/utils.h"

namespace caffe2 {
namespace math {

namespace {

#define DELEGATE_SIMPLE_CUDA_UNARY_KERNEL_FUNCTION(T, Func, DeviceFunc) \
  __global__ void Func##CUDAKernel(const int N, const T* X, T* Y) {     \
    const int i = blockIdx.x * CAFFE_CUDA_NUM_THREADS + threadIdx.x;    \
    if (i < N) {                                                        \
      Y[i] = DeviceFunc(X[i]);                                          \
    }                                                                   \
  }
DELEGATE_SIMPLE_CUDA_UNARY_KERNEL_FUNCTION(float, Exp, expf)
DELEGATE_SIMPLE_CUDA_UNARY_KERNEL_FUNCTION(float, Log, logf)
DELEGATE_SIMPLE_CUDA_UNARY_KERNEL_FUNCTION(float, Cos, cosf)
DELEGATE_SIMPLE_CUDA_UNARY_KERNEL_FUNCTION(float, Acos, acosf)
DELEGATE_SIMPLE_CUDA_UNARY_KERNEL_FUNCTION(float, Sin, sinf)
DELEGATE_SIMPLE_CUDA_UNARY_KERNEL_FUNCTION(float, Asin, asinf)
DELEGATE_SIMPLE_CUDA_UNARY_KERNEL_FUNCTION(float, Tan, tanf)
DELEGATE_SIMPLE_CUDA_UNARY_KERNEL_FUNCTION(float, Atan, atanf)
DELEGATE_SIMPLE_CUDA_UNARY_KERNEL_FUNCTION(float, Sinh, sinhf)
DELEGATE_SIMPLE_CUDA_UNARY_KERNEL_FUNCTION(float, Cosh, coshf)
DELEGATE_SIMPLE_CUDA_UNARY_KERNEL_FUNCTION(float, Tanh, tanhf)
DELEGATE_SIMPLE_CUDA_UNARY_KERNEL_FUNCTION(float, Abs, fabsf)
DELEGATE_SIMPLE_CUDA_UNARY_KERNEL_FUNCTION(float, Sqr, utils::Square<float>)
DELEGATE_SIMPLE_CUDA_UNARY_KERNEL_FUNCTION(float, Sqrt, sqrtf)
DELEGATE_SIMPLE_CUDA_UNARY_KERNEL_FUNCTION(float, Rsqrt, rsqrtf)
DELEGATE_SIMPLE_CUDA_UNARY_KERNEL_FUNCTION(float, Cbrt, cbrtf)
DELEGATE_SIMPLE_CUDA_UNARY_KERNEL_FUNCTION(float, Erf, erff)
DELEGATE_SIMPLE_CUDA_UNARY_KERNEL_FUNCTION(double, Erf, erf)
DELEGATE_SIMPLE_CUDA_UNARY_KERNEL_FUNCTION(
    std::int32_t,
    Cube,
    utils::Cube<std::int32_t>)
DELEGATE_SIMPLE_CUDA_UNARY_KERNEL_FUNCTION(
    std::int64_t,
    Cube,
    utils::Cube<std::int64_t>)
DELEGATE_SIMPLE_CUDA_UNARY_KERNEL_FUNCTION(float, Cube, utils::Cube<float>)
DELEGATE_SIMPLE_CUDA_UNARY_KERNEL_FUNCTION(double, Cube, utils::Cube<double>)
DELEGATE_SIMPLE_CUDA_UNARY_KERNEL_FUNCTION(bool, Not, utils::Not<bool>)
DELEGATE_SIMPLE_CUDA_UNARY_KERNEL_FUNCTION(
    std::int32_t,
    Neg,
    utils::Negate<std::int32_t>)
DELEGATE_SIMPLE_CUDA_UNARY_KERNEL_FUNCTION(
    std::int64_t,
    Neg,
    utils::Negate<std::int64_t>)
DELEGATE_SIMPLE_CUDA_UNARY_KERNEL_FUNCTION(float, Neg, utils::Negate<float>)
DELEGATE_SIMPLE_CUDA_UNARY_KERNEL_FUNCTION(double, Neg, utils::Negate<double>)
DELEGATE_SIMPLE_CUDA_UNARY_KERNEL_FUNCTION(
    std::int32_t,
    Sign,
    utils::Sign<std::int32_t>)
DELEGATE_SIMPLE_CUDA_UNARY_KERNEL_FUNCTION(
    std::int64_t,
    Sign,
    utils::Sign<std::int64_t>)
DELEGATE_SIMPLE_CUDA_UNARY_KERNEL_FUNCTION(float, Sign, utils::Sign<float>)
DELEGATE_SIMPLE_CUDA_UNARY_KERNEL_FUNCTION(double, Sign, utils::Sign<double>)
DELEGATE_SIMPLE_CUDA_UNARY_KERNEL_FUNCTION(float, Inv, utils::Inv<float>)
DELEGATE_SIMPLE_CUDA_UNARY_KERNEL_FUNCTION(double, Inv, utils::Inv<double>)
#undef DELEGATE_SIMPLE_CUDA_UNARY_KERNEL_FUNCTION

template <typename T>
__global__ void SinCosCUDAKernel(const int N, const T* X, T* S, T* C) {
  const int i = blockIdx.x * CAFFE_CUDA_NUM_THREADS + threadIdx.x;
  if (i < N) {
#if __CUDA_ARCH__ >= 350 || defined(__HIP_PLATFORM_HCC__)
    c10::cuda::compat::sincos(__ldg(X + i), S + i, C + i);
#else
    c10::cuda::compat::sincos(X[i], S + i, C + i);
#endif
  }
}

template <typename T, class Func>
__global__ void SimpleBinaryCUDAKernel(
    const int N,
    const Func func,
    const T* A,
    const T* B,
    T* C) {
  const int i = blockIdx.x * CAFFE_CUDA_NUM_THREADS + threadIdx.x;
  if (i < N) {
    C[i] = func(A[i], B[i]);
  }
}

template <typename T, class Comp>
__global__ void SimpleCompareCUDAKernel(
    const int N,
    const Comp comp,
    const T* A,
    const T* B,
    bool* C) {
  const int i = blockIdx.x * CAFFE_CUDA_NUM_THREADS + threadIdx.x;
  if (i < N) {
    C[i] = comp(A[i], B[i]);
  }
}

} // namespace

#define DEFINE_SIMPLE_CUDA_UNARY_FUNCTION(T, Func)           \
  template <>                                                \
  CAFFE2_CUDA_EXPORT void Func<T, HIPContext>(              \
      const int N, const T* X, T* Y, HIPContext* context) { \
    if (N > 0) {                                             \
      const int M = DivUp(N, CAFFE_CUDA_NUM_THREADS);        \
      Func##CUDAKernel<<<                                    \
          M,                                                 \
          CAFFE_CUDA_NUM_THREADS,                            \
          0,                                                 \
          context->hip_stream()>>>(N, X, Y);                \
    }                                                        \
  }
DEFINE_SIMPLE_CUDA_UNARY_FUNCTION(float, Exp)
DEFINE_SIMPLE_CUDA_UNARY_FUNCTION(float, Log)
DEFINE_SIMPLE_CUDA_UNARY_FUNCTION(float, Cos)
DEFINE_SIMPLE_CUDA_UNARY_FUNCTION(float, Acos)
DEFINE_SIMPLE_CUDA_UNARY_FUNCTION(float, Sin)
DEFINE_SIMPLE_CUDA_UNARY_FUNCTION(float, Asin)
DEFINE_SIMPLE_CUDA_UNARY_FUNCTION(float, Tan)
DEFINE_SIMPLE_CUDA_UNARY_FUNCTION(float, Atan)
DEFINE_SIMPLE_CUDA_UNARY_FUNCTION(float, Sinh)
DEFINE_SIMPLE_CUDA_UNARY_FUNCTION(float, Cosh)
DEFINE_SIMPLE_CUDA_UNARY_FUNCTION(float, Tanh)
DEFINE_SIMPLE_CUDA_UNARY_FUNCTION(float, Abs)
DEFINE_SIMPLE_CUDA_UNARY_FUNCTION(float, Sqr)
DEFINE_SIMPLE_CUDA_UNARY_FUNCTION(float, Sqrt)
DEFINE_SIMPLE_CUDA_UNARY_FUNCTION(float, Rsqrt)
DEFINE_SIMPLE_CUDA_UNARY_FUNCTION(float, Cbrt)
DEFINE_SIMPLE_CUDA_UNARY_FUNCTION(float, Erf)
DEFINE_SIMPLE_CUDA_UNARY_FUNCTION(double, Erf)
DEFINE_SIMPLE_CUDA_UNARY_FUNCTION(float, Cube)
DEFINE_SIMPLE_CUDA_UNARY_FUNCTION(double, Cube)
DEFINE_SIMPLE_CUDA_UNARY_FUNCTION(std::int32_t, Cube)
DEFINE_SIMPLE_CUDA_UNARY_FUNCTION(std::int64_t, Cube)
DEFINE_SIMPLE_CUDA_UNARY_FUNCTION(bool, Not)
DEFINE_SIMPLE_CUDA_UNARY_FUNCTION(float, Neg)
DEFINE_SIMPLE_CUDA_UNARY_FUNCTION(double, Neg)
DEFINE_SIMPLE_CUDA_UNARY_FUNCTION(std::int32_t, Neg)
DEFINE_SIMPLE_CUDA_UNARY_FUNCTION(std::int64_t, Neg)
DEFINE_SIMPLE_CUDA_UNARY_FUNCTION(float, Sign)
DEFINE_SIMPLE_CUDA_UNARY_FUNCTION(double, Sign)
DEFINE_SIMPLE_CUDA_UNARY_FUNCTION(std::int32_t, Sign)
DEFINE_SIMPLE_CUDA_UNARY_FUNCTION(std::int64_t, Sign)
DEFINE_SIMPLE_CUDA_UNARY_FUNCTION(float, Inv)
DEFINE_SIMPLE_CUDA_UNARY_FUNCTION(double, Inv)
#undef DEFINE_SIMPLE_CUDA_UNARY_FUNCTION

#define CAFFE2_SPECIALIZED_CUDA_SINCOS(T)                             \
  template <>                                                         \
  CAFFE2_CUDA_EXPORT void SinCos<T, HIPContext>(                     \
      const int N, const T* X, T* S, T* C, HIPContext* context) {    \
    if (N > 0) {                                                      \
      const int K = DivUp(N, CAFFE_CUDA_NUM_THREADS);                 \
      SinCosCUDAKernel<T>                                             \
          <<<K, CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>( \
              N, X, S, C);                                            \
    }                                                                 \
  }
CAFFE2_SPECIALIZED_CUDA_SINCOS(float)
CAFFE2_SPECIALIZED_CUDA_SINCOS(double)
#undef CAFFE2_SPECIALIZED_CUDA_SINCOS

#define DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(T, Func, DeviceFunc)        \
  template <>                                                            \
  CAFFE2_CUDA_EXPORT void Func<T, HIPContext>(                          \
      const int N, const T* A, const T* B, T* C, HIPContext* context) { \
    if (N > 0) {                                                         \
      const int M = DivUp(N, CAFFE_CUDA_NUM_THREADS);                    \
      SimpleBinaryCUDAKernel<<<                                          \
          M,                                                             \
          CAFFE_CUDA_NUM_THREADS,                                        \
          0,                                                             \
          context->hip_stream()>>>(N, DeviceFunc, A, B, C);             \
    }                                                                    \
  }
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(
    std::int32_t,
    Add,
    thrust::plus<std::int32_t>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(
    std::int64_t,
    Add,
    thrust::plus<std::int64_t>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(float, Add, thrust::plus<float>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(double, Add, thrust::plus<double>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(at::Half, Add, utils::HalfAddFunctor())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(
    std::int32_t,
    Sub,
    thrust::minus<std::int32_t>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(
    std::int64_t,
    Sub,
    thrust::minus<std::int64_t>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(float, Sub, thrust::minus<float>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(double, Sub, thrust::minus<double>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(at::Half, Sub, utils::HalfSubFunctor())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(
    std::int32_t,
    Mul,
    thrust::multiplies<std::int32_t>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(
    std::int64_t,
    Mul,
    thrust::multiplies<std::int64_t>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(float, Mul, thrust::multiplies<float>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(double, Mul, thrust::multiplies<double>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(at::Half, Mul, utils::HalfMulFunctor())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(
    std::int32_t,
    Div,
    thrust::divides<std::int32_t>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(
    std::int64_t,
    Div,
    thrust::divides<std::int64_t>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(float, Div, thrust::divides<float>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(double, Div, thrust::divides<double>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(at::Half, Div, utils::HalfDivFunctor())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(float, Min, thrust::minimum<float>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(double, Min, thrust::minimum<double>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(float, Max, thrust::maximum<float>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(double, Max, thrust::maximum<double>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(bool, And, thrust::logical_and<bool>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(bool, Or, thrust::logical_or<bool>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(bool, Xor, thrust::bit_xor<bool>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(bool, BitwiseAnd, thrust::bit_and<bool>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(
    std::int32_t,
    BitwiseAnd,
    thrust::bit_and<std::int32_t>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(
    std::int64_t,
    BitwiseAnd,
    thrust::bit_and<std::int64_t>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(bool, BitwiseOr, thrust::bit_or<bool>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(
    std::int32_t,
    BitwiseOr,
    thrust::bit_or<std::int32_t>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(
    std::int64_t,
    BitwiseOr,
    thrust::bit_or<std::int64_t>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(bool, BitwiseXor, thrust::bit_xor<bool>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(
    std::int32_t,
    BitwiseXor,
    thrust::bit_xor<std::int32_t>())
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(
    std::int64_t,
    BitwiseXor,
    thrust::bit_xor<std::int64_t>())
#undef DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION

#define DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(T, Func, DeviceComp)          \
  template <>                                                               \
  CAFFE2_CUDA_EXPORT void Func<T, HIPContext>(                             \
      const int N, const T* A, const T* B, bool* C, HIPContext* context) { \
    if (N > 0) {                                                            \
      const int M = DivUp(N, CAFFE_CUDA_NUM_THREADS);                       \
      SimpleCompareCUDAKernel<<<                                            \
          M,                                                                \
          CAFFE_CUDA_NUM_THREADS,                                           \
          0,                                                                \
          context->hip_stream()>>>(N, DeviceComp, A, B, C);                \
    }                                                                       \
  }
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(bool, EQ, thrust::equal_to<bool>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(
    std::int32_t,
    EQ,
    thrust::equal_to<std::int32_t>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(
    std::int64_t,
    EQ,
    thrust::equal_to<std::int64_t>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(float, EQ, thrust::equal_to<float>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(double, EQ, thrust::equal_to<double>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(bool, NE, thrust::not_equal_to<bool>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(
    std::int32_t,
    NE,
    thrust::not_equal_to<std::int32_t>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(
    std::int64_t,
    NE,
    thrust::not_equal_to<std::int64_t>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(float, NE, thrust::not_equal_to<float>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(
    double,
    NE,
    thrust::not_equal_to<double>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(bool, LT, thrust::less<bool>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(
    std::int32_t,
    LT,
    thrust::less<std::int32_t>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(
    std::int64_t,
    LT,
    thrust::less<std::int64_t>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(float, LT, thrust::less<float>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(double, LT, thrust::less<double>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(bool, LE, thrust::less_equal<bool>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(
    std::int32_t,
    LE,
    thrust::less_equal<std::int32_t>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(
    std::int64_t,
    LE,
    thrust::less_equal<std::int64_t>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(float, LE, thrust::less_equal<float>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(double, LE, thrust::less_equal<double>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(bool, GT, thrust::greater<bool>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(
    std::int32_t,
    GT,
    thrust::greater<std::int32_t>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(
    std::int64_t,
    GT,
    thrust::greater<std::int64_t>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(float, GT, thrust::greater<float>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(double, GT, thrust::greater<double>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(bool, GE, thrust::greater_equal<bool>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(
    std::int32_t,
    GE,
    thrust::greater_equal<std::int32_t>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(
    std::int64_t,
    GE,
    thrust::greater_equal<std::int64_t>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(float, GE, thrust::greater_equal<float>())
DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION(
    double,
    GE,
    thrust::greater_equal<double>())
#undef DELEGATE_SIMPLE_CUDA_COMPARE_FUNCTION

} // namespace math
} // namespace caffe2
