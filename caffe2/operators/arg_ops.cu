#include "hip/hip_runtime.h"
#include "caffe2/operators/arg_ops.h"

#include <limits>

#include <cub/block/block_reduce.cuh>
#include <hipcub/hipcub.hpp>

#include "caffe2/core/common_gpu.h"
#include "caffe2/core/context_gpu.h"

namespace caffe2 {

namespace {

template <typename T>
using KeyValuePair = hipcub::KeyValuePair<TIndex, T>;

template <typename T>
using BlockReduce = hipcub::BlockReduce<KeyValuePair<T>, CAFFE_CUDA_NUM_THREADS>;

template <typename T, class ReduceOp>
__global__ void ComputeArgCUDAKernel(
    const T* X,
    const TIndex outer_size,
    const TIndex inner_size,
    const TIndex stride,
    const ReduceOp& reduce_op,
    const T init,
    TIndex* Y) {
  __shared__ typename BlockReduce<T>::TempStorage temp_storage;
  for (TIndex idx = blockIdx.x; idx < outer_size; idx += gridDim.x) {
    const TIndex i = idx / stride;
    const TIndex j = idx % stride;
    KeyValuePair<T> kv = {-1, init};
    for (TIndex k = threadIdx.x; k < inner_size; k += blockDim.x) {
      kv = reduce_op({k, X[i * inner_size * stride + k * stride + j]}, kv);
    }
    kv = BlockReduce<T>(temp_storage).Reduce(kv, reduce_op);
    if (threadIdx.x == 0) {
      Y[idx] = kv.key;
    }
    __syncthreads();
  }
}

} // namespace

template <typename T, typename Context>
class ArgMaxCudaOp final : public ArgOpBase<T, Context> {
 public:
  USE_OPERATOR_FUNCTIONS(Context);

  ArgMaxCudaOp(const OperatorDef& operator_def, Workspace* ws)
      : ArgOpBase<T, Context>(operator_def, ws) {}

 protected:
  bool Compute(
      const T* X,
      const TIndex prev_size,
      const TIndex next_size,
      const TIndex n,
      TIndex* Y) override;
};

template <typename T, typename Context>
bool ArgMaxCudaOp<T, Context>::Compute(
    const T* X,
    const TIndex prev_size,
    const TIndex next_size,
    const TIndex n,
    TIndex* Y) {
  const TIndex outer_size = prev_size * next_size;
  ComputeArgCUDAKernel<<<
      std::min(outer_size, static_cast<TIndex>(CAFFE_MAXIMUM_NUM_BLOCKS)),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      X,
      outer_size,
      n,
      next_size,
      hipcub::ArgMax(),
      std::numeric_limits<T>::lowest(),
      Y);
  return true;
}

template <typename T, typename Context>
class ArgMinCudaOp final : public ArgOpBase<T, Context> {
 public:
  USE_OPERATOR_FUNCTIONS(Context);

  ArgMinCudaOp(const OperatorDef& operator_def, Workspace* ws)
      : ArgOpBase<T, Context>(operator_def, ws) {}

 protected:
  bool Compute(
      const T* X,
      const TIndex prev_size,
      const TIndex next_size,
      const TIndex n,
      TIndex* Y) override;
};

template <typename T, typename Context>
bool ArgMinCudaOp<T, Context>::Compute(
    const T* X,
    const TIndex prev_size,
    const TIndex next_size,
    const TIndex n,
    TIndex* Y) {
  const TIndex outer_size = prev_size * next_size;
  ComputeArgCUDAKernel<<<
      std::min(outer_size, static_cast<TIndex>(CAFFE_MAXIMUM_NUM_BLOCKS)),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      X,
      outer_size,
      n,
      next_size,
      hipcub::ArgMin(),
      std::numeric_limits<T>::max(),
      Y);
  return true;
}

REGISTER_HIP_OPERATOR(ArgMax, ArgMaxCudaOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(ArgMin, ArgMinCudaOp<float, HIPContext>);

} // namespace caffe2
