#include "hip/hip_runtime.h"
#include <cub/block/block_reduce.cuh>

#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/tile_op.h"

namespace caffe2 {
namespace {
template <typename T>
__global__ void TileCopyKernel(
    int outer_dim,
    int inner_dim,
    int tiles,
    const T* input_data,
    T* output_data) {
  CUDA_1D_KERNEL_LOOP(index, outer_dim * inner_dim * tiles) {
    int col = index % inner_dim;
    int row = index / (inner_dim * tiles);
    output_data[index] = input_data[row * inner_dim + col];
  }
}

template <typename T>
__global__ void TileGradientAxpyKernel(
    int outer_dim,
    int inner_dim,
    int tiles,
    const T* input_data,
    T* output_data) {
  typedef hipcub::BlockReduce<T, CAFFE_CUDA_NUM_THREADS> BlockReduce;

  for (int idx = blockIdx.x; idx < outer_dim * inner_dim; idx += gridDim.x) {
    int i = idx / inner_dim;
    int j = idx % inner_dim;
    T* output_ptr = output_data + inner_dim * i;

    T x = 0.0;
    for (int t = threadIdx.x; t < tiles; t += blockDim.x) {
      const T* input_ptr = input_data + (i * tiles + t) * inner_dim;
      x += input_ptr[j];
    }
    __shared__ typename BlockReduce::TempStorage temp_storage;
    T totx = BlockReduce(temp_storage).Sum(x);
    if (threadIdx.x == 0) {
      output_ptr[j] = totx;
    }
    __syncthreads();
  }
}
} // namespace

template <>
void TileOp<HIPContext>::DoTile(
    const TypeMeta& meta,
    int item_size,
    int outer_dim,
    int inner_dim,
    const char* input_data,
    char* output_data) {
  TileCopyKernel<float>
      <<<std::min(outer_dim * inner_dim * tiles_, CAFFE_MAXIMUM_NUM_BLOCKS),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context_.hip_stream()>>>(
          outer_dim,
          inner_dim,
          tiles_,
          reinterpret_cast<const float*>(input_data),
          reinterpret_cast<float*>(output_data));
}

template <>
void TileGradientOp<float, HIPContext>::DoTileGradient(
    const TypeMeta& meta,
    int item_size,
    int outer_dim,
    int inner_dim,
    const char* input_data,
    char* output_data) {
  TileGradientAxpyKernel<float><<<
      std::min(outer_dim * inner_dim, CAFFE_MAXIMUM_NUM_BLOCKS),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context_.hip_stream()>>>(
      outer_dim,
      inner_dim,
      tiles_,
      reinterpret_cast<const float*>(input_data),
      reinterpret_cast<float*>(output_data));
}

REGISTER_HIP_OPERATOR(Tile, TileOp<HIPContext>);
REGISTER_HIP_OPERATOR(TileGradient, TileGradientOp<float, HIPContext>);
} // namespace caffe2
