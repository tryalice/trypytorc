#include "hip/hip_runtime.h"
#include "caffe2/operators/elementwise_div_op.h"

#include <algorithm>
#include <functional>

#include <cub/block/block_reduce.cuh>
#include <hipcub/hipcub.hpp>

#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/elementwise_ops_utils.h"

namespace caffe2 {

namespace {

template <typename T>
using BlockReduce = hipcub::BlockReduce<T, CAFFE_CUDA_NUM_THREADS>;

template <typename TGrad, typename TIn, int D>
__global__ void ComputeDivAGradientCUDAKernel(
    const int outer_size,
    const int inner_size,
    const SimpleArray<int, D> C_dims,
    const SimpleArray<int, D> C_strides,
    const SimpleArray<int, D> B_strides,
    const SimpleArray<int, D> A_dims,
    const TGrad* dC,
    const TIn* B,
    TGrad* dA) {
  __shared__ typename BlockReduce<TGrad>::TempStorage temp_storage;
  for (int i = blockIdx.x; i < outer_size; i += gridDim.x) {
    TGrad sum = 0;
    for (int j = threadIdx.x; j < inner_size; j += blockDim.x) {
      const int A_index = i * inner_size + j;
      int C_index = 0;
      int A_index_val = A_index;
#pragma unroll
      for (int d = D - 1; d >= 0; --d) {
        C_index += (A_index_val % A_dims.data[d]) * C_strides.data[d];
        A_index_val /= A_dims.data[d];
      }
      int B_index = 0;
      int C_index_val = C_index;
#pragma unroll
      for (int d = D - 1; d >= 0; --d) {
        B_index += B_strides.data[d] == 0
            ? 0
            : (C_index_val % C_dims.data[d]) * B_strides.data[d];
        C_index_val /= C_dims.data[d];
      }
#if __CUDA_ARCH__ >= 350
      sum += __ldg(dC + C_index) / __ldg(B + B_index);
#else
      sum += dC[C_index] / B[B_index];
#endif
    }
    sum = BlockReduce<TGrad>(temp_storage).Reduce(sum, hipcub::Sum());
    if (threadIdx.x == 0) {
      dA[i] = sum;
    }
    __syncthreads();
  }
}

template <typename TGrad, typename TIn, typename TOut>
__global__ void ComputeSimpleDivBGradientCUDAKernel(
    const int size,
    const TGrad* dC,
    const TIn* B,
    const TOut* C,
    TGrad* dB) {
  CUDA_1D_KERNEL_LOOP(i, size) {
#if __CUDA_ARCH__ >= 350
    dB[i] = -__ldg(dC + i) * __ldg(C + i) / __ldg(B + i);
#else
    dB[i] = -dC[i] * C[i] / B[i];
#endif
  }
}

template <typename TGrad, typename TIn, typename TOut, int D>
__global__ void ComputeDivBGradientCUDAKernel(
    const int outer_size,
    const int inner_size,
    const SimpleArray<int, D> C_strides,
    const SimpleArray<int, D> B_dims,
    const TGrad* dC,
    const TIn* B,
    const TOut* C,
    TGrad* dB) {
  __shared__ typename BlockReduce<TGrad>::TempStorage temp_storage;
  for (int i = blockIdx.x; i < outer_size; i += gridDim.x) {
    TGrad sum = 0;
    for (int j = threadIdx.x; j < inner_size; j += blockDim.x) {
      int C_index = 0;
      int B_index = i * inner_size + j;
#pragma unroll
      for (int d = D - 1; d >= 0; --d) {
        C_index += (B_index % B_dims.data[d]) * C_strides.data[d];
        B_index /= B_dims.data[d];
      }
#if __CUDA_ARCH__ >= 350
      sum += -__ldg(dC + C_index) * __ldg(C + C_index) / __ldg(B + i);
#else
      sum += -dC[C_index] * C[C_index] / B[i];
#endif
    }
    sum = BlockReduce<TGrad>(temp_storage).Reduce(sum, hipcub::Sum());
    if (threadIdx.x == 0) {
      dB[i] = sum;
    }
    __syncthreads();
  }
}

template <typename TGrad, typename TIn, int D>
void ComputeDivAGradientCUDAImpl(
    const int outer_size,
    const int inner_size,
    const int* C_dims,
    const int* B_dims,
    const int* A_axes,
    const TGrad* dC,
    const TIn* B,
    TGrad* dA,
    HIPContext* context) {
  SimpleArray<int, D> C_dims_arr;
  SimpleArray<int, D> C_strides_arr;
  SimpleArray<int, D> B_strides_arr;
  SimpleArray<int, D> A_dims_arr;
  std::copy_n(C_dims, D, C_dims_arr.data);
  math::utils::ComputeTransposedStrides(D, C_dims, A_axes, C_strides_arr.data);
  int cur_stride = 1;
  for (int i = D - 1; i >= 0; --i) {
    B_strides_arr.data[i] = B_dims[i] == 1 ? 0 : cur_stride;
    cur_stride *= B_dims[i];
  }
  for (int i = 0; i < D; ++i) {
    A_dims_arr.data[i] = C_dims[A_axes[i]];
  }
  ComputeDivAGradientCUDAKernel<TGrad, TIn, D>
      <<<std::min(outer_size, CAFFE_MAXIMUM_NUM_BLOCKS),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context->hip_stream()>>>(
          outer_size,
          inner_size,
          C_dims_arr,
          C_strides_arr,
          B_strides_arr,
          A_dims_arr,
          dC,
          B,
          dA);
}

template <typename TGrad, typename TIn, typename TOut, int D>
void ComputeDivBGradientCUDAImpl(
    const int outer_size,
    const int inner_size,
    const int* C_dims,
    const int* B_axes,
    const TGrad* dC,
    const TIn* B,
    const TOut* C,
    TGrad* dB,
    HIPContext* context) {
  SimpleArray<int, D> C_strides_arr;
  SimpleArray<int, D> B_dims_arr;
  math::utils::ComputeTransposedStrides(D, C_dims, B_axes, C_strides_arr.data);
  for (int i = 0; i < D; ++i) {
    B_dims_arr.data[i] = C_dims[B_axes[i]];
  }
  ComputeDivBGradientCUDAKernel<TGrad, TIn, TOut, D>
      <<<std::min(outer_size, CAFFE_MAXIMUM_NUM_BLOCKS),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context->hip_stream()>>>(
          outer_size, inner_size, C_strides_arr, B_dims_arr, dC, B, C, dB);
}

template <typename TGrad, typename TIn>
void ComputeDivAGradientCUDA(
    const std::vector<int>& C_dims,
    const std::vector<int>& B_dims,
    const std::vector<int>& A_axes,
    const TGrad* dC,
    const TIn* B,
    TGrad* dA,
    HIPContext* context) {
  CAFFE_ENFORCE_EQ(C_dims.size(), B_dims.size());
  const int ndim = C_dims.size();
  std::vector<int> A_transpose_axes(ndim);
  math::utils::ComputeTransposeAxesForReduceOp(
      ndim, A_axes.size(), A_axes.data(), A_transpose_axes.data());
  const int pivot = ndim - A_axes.size();
  int outer_size = 1;
  for (int i = 0; i < pivot; ++i) {
    outer_size *= C_dims[A_transpose_axes[i]];
  }
  int inner_size = 1;
  for (int i = pivot; i < ndim; ++i) {
    inner_size *= C_dims[A_transpose_axes[i]];
  }
  DISPATCH_FUNCTION_BY_VALUE_WITH_TYPE_2(
      ndim,
      ComputeDivAGradientCUDAImpl,
      TGrad,
      TIn,
      outer_size,
      inner_size,
      C_dims.data(),
      B_dims.data(),
      A_transpose_axes.data(),
      dC,
      B,
      dA,
      context);
}

template <typename TGrad, typename TIn, typename TOut>
void ComputeDivBGradientCUDA(
    const std::vector<int>& C_dims,
    const std::vector<int>& B_axes,
    const TGrad* dC,
    const TIn* B,
    const TOut* C,
    TGrad* dB,
    HIPContext* context) {
  const int ndim = C_dims.size();
  std::vector<int> B_transpose_axes(ndim);
  math::utils::ComputeTransposeAxesForReduceOp(
      ndim, B_axes.size(), B_axes.data(), B_transpose_axes.data());
  const int pivot = ndim - B_axes.size();
  int outer_size = 1;
  for (int i = 0; i < pivot; ++i) {
    outer_size *= C_dims[B_transpose_axes[i]];
  }
  int inner_size = 1;
  for (int i = pivot; i < ndim; ++i) {
    inner_size *= C_dims[B_transpose_axes[i]];
  }
  DISPATCH_FUNCTION_BY_VALUE_WITH_TYPE_3(
      ndim,
      ComputeDivBGradientCUDAImpl,
      TGrad,
      TIn,
      TOut,
      outer_size,
      inner_size,
      C_dims.data(),
      B_transpose_axes.data(),
      dC,
      B,
      C,
      dB,
      context);
}

} // namespace

template <>
template <typename TGrad, typename TIn, typename TOut>
bool DivFunctor<HIPContext>::Backward(
    const std::vector<int>& A_dims,
    const std::vector<int>& B_dims,
    const TGrad* dC,
    const TIn* /* A */,
    const TIn* B,
    const TOut* C,
    TGrad* dA,
    TGrad* dB,
    HIPContext* context) const {
  if (A_dims == B_dims) {
    const int size = std::accumulate(
        A_dims.cbegin(), A_dims.cend(), 1, std::multiplies<int>());
    math::Div(size, dC, B, dA, context);
    ComputeSimpleDivBGradientCUDAKernel<TGrad, TIn, TOut>
        <<<CAFFE_GET_BLOCKS(size),
           CAFFE_CUDA_NUM_THREADS,
           0,
           context->hip_stream()>>>(size, dC, B, C, dB);
    return true;
  }
  const int ndim = std::max(A_dims.size(), B_dims.size());
  std::vector<int> A_broadcast_dims(ndim);
  std::vector<int> B_broadcast_dims(ndim);
  std::vector<int> C_broadcast_dims(ndim);
  math::utils::ComputeBroadcastBinaryOpDims(
      A_dims.size(),
      A_dims.data(),
      B_dims.size(),
      B_dims.data(),
      A_broadcast_dims.data(),
      B_broadcast_dims.data(),
      C_broadcast_dims.data());
  std::vector<int> A_axes;
  std::vector<int> B_axes;
  elementwise_ops_utils::ComputeBinaryBroadcastBackwardAxes(
      A_dims, B_dims, &A_axes, &B_axes);
  ComputeDivAGradientCUDA<TGrad, TIn>(
      C_broadcast_dims, B_broadcast_dims, A_axes, dC, B, dA, context);
  ComputeDivBGradientCUDA<TGrad, TIn, TOut>(
      C_broadcast_dims, B_axes, dC, B, C, dB, context);
  return true;
}

REGISTER_HIP_OPERATOR(
    Div,
    BinaryElementwiseOp<NumericTypes, HIPContext, DivFunctor<HIPContext>>);
REGISTER_HIP_OPERATOR(
    DivGradient,
    BinaryElementwiseGradientOp<
        NumericTypes,
        HIPContext,
        DivFunctor<HIPContext>>);

} // namespace caffe2
