#include "hip/hip_runtime.h"
#include "caffe2/operators/rsqrt_op.h"

#include <algorithm>
#include <functional>

#include "caffe2/core/context_gpu.h"

namespace caffe2 {

namespace {

template <typename T>
inline __host__ __device__ T CubeCUDA(const T x) {
  return x * x * x;
}

template <typename T>
__global__ void
RSqrtGradientCUDAKernel(const int size, const T* dY, const T* Y, T* dX) {
  CUDA_1D_KERNEL_LOOP(i, size) {
#if __CUDA_ARCH__ >= 350
    dX[i] = __ldg(dY + i) * CubeCUDA(__ldg(Y + i)) * static_cast<T>(-0.5);
#else
    dX[i] = dY[i] * CubeCUDA(Y[i]) * static_cast<T>(-0.5);
#endif
  }
}

} // namespace

template <>
template <typename T>
bool RSqrtGradientFunctor<HIPContext>::Forward(
    const std::vector<int>& dY_dims,
    const std::vector<int>& /* Y_dims */,
    const T* dY,
    const T* Y,
    T* dX,
    HIPContext* context) const {
  const int size = std::accumulate(
      dY_dims.cbegin(), dY_dims.cend(), 1, std::multiplies<int>());
  RSqrtGradientCUDAKernel<T>
      <<<CAFFE_GET_BLOCKS(size),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context->hip_stream()>>>(size, dY, Y, dX);
  return true;
}

REGISTER_HIP_OPERATOR(
    RSqrt,
    UnaryElementwiseOp<
        TensorTypes<float>,
        HIPContext,
        RSqrtFunctor<HIPContext>>);
REGISTER_HIP_OPERATOR(
    RSqrtGradient,
    BinaryElementwiseOp<
        TensorTypes<float>,
        HIPContext,
        RSqrtGradientFunctor<HIPContext>>);

} // namespace caffe2
